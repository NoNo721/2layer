#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cmath>
#include <fstream>
#include <string.h>
#include <unistd.h>
#include <sys/stat.h>

using namespace std;

void S_layer1_up(double S[])
{
	char fileName[100];
	sprintf(fileName,"./Gx/xxx.txt");

	ifstream fin_gx(fileName);
	if(!(fin_gx))
	{
		printf("the %s is set up.\n",fileName);

		ofstream fout_gx(fileName);

		double *sum,*d_x;
    	double dt;
    	int n_steps = 1e5;

    	hipMallocManaged(&sum, n * n_steps * sizeof(double));
    	hipMalloc((void **)&d_x, (n + 1) * sizeof(double));

    	hipMemcpyAsync(d_x, x, (n + 1) * sizeof(double), hipMemcpyHostToDevice);

    	dt=(Up-Low)/n_steps;
    
    	int blockSize = 256;
    	int numBlock = (n * n_steps + blockSize - 1) / blockSize;

    	S_integrand<<<numBlock, blockSize>>>(n, n_steps, g_A, Low, dt, sum, d_x, R, injectionmode);
        
    	hipDeviceSynchronize();
    
    	
    	for(int i = 1; i <= n; i++)
    	{
    		S[i]=0.0;
        	for(int j=0; j < n_steps; j++)
				S[i] += sum[(i - 1) * n_steps + j];
			S[i] = S[i]/cq;
			fout_gx<<S[i]<<endl;
			procBar(i*100/n);
		}
		cout<<endl;
		
    	hipFree(sum);
    	hipFree(d_x);

    	fout_gx.close();
	}
	else
	{
		for(int i=1; i<=n; i++)
			fin_gx>>S[i];
		fin_gx.close();
	}
	cout<<"<S> done...\n"<<endl;
}