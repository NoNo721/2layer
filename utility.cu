
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cmath>
#include <fstream>
#include <string.h>
#include <unistd.h>
#include <sys/stat.h>

using namespace std;

void procBar(int rate)
//进度条
{
	int ratep = rate/2;
	char bar[52];		//51个“=”，一个"\0"
	const char *label = "|/-\\";

	memset(bar, 0, 52*sizeof(char));
	
	int i = 0;
	while(i <= ratep)
	{
		bar[i] = '=';
		i++;
	}
	
	fflush(stdout);	
	printf("Proc:[%-51s][%d%%][%c]\r", bar, rate, label[rate%4]);	//"-"代表左对齐，‘\r’回车（即光标位于行首）
}

void myMkdir(const char *path)
//判断文件夹是否存在，不存在就创建
{
	 if(access(path,0)==-1)
        {
           printf("%s is set up.\n",path);
           mkdir(path,0777);
        }
}









